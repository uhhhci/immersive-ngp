#include "hip/hip_runtime.h"
#include "neural-graphics-primitives/unity.h"
#include "Unity/IUnityInterface.h"
#include "Unity/IUnityGraphics.h"

#ifdef _WIN32
#  include <GL/gl3w.h>
#else
#  include <GL/glew.h>
#endif
#include <GLFW/glfw3.h>
#include "gl/GL.h"
#include "gl/GLU.h"

#include <neural-graphics-primitives/adam_optimizer.h>
#include <neural-graphics-primitives/camera_path.h>
#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/discrete_distribution.h>
#include <neural-graphics-primitives/nerf.h>
#include <neural-graphics-primitives/nerf_loader.h>
#include <neural-graphics-primitives/render_buffer.h>
#include <neural-graphics-primitives/sdf.h>
#include <neural-graphics-primitives/shared_queue.h>
#include <neural-graphics-primitives/trainable_buffer.cuh>
#include <neural-graphics-primitives/testbed.h>
#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/render_buffer.h>
#include <neural-graphics-primitives/tinyexr_wrapper.h>

#include <tiny-cuda-nn/gpu_memory.h>
#include <filesystem/path.h>
#include <cuda_gl_interop.h>

#include <tiny-cuda-nn/multi_stream.h>
#include <tiny-cuda-nn/random.h>

#include <json/json.hpp>
#include <filesystem/path.h>
#include <thread>
#include "gl/GL.h"
#include "gl/GLU.h"
#include <memory>

using Texture = std::shared_ptr<ngp::GLTexture>;
using RenderBuffer = std::shared_ptr<ngp::CudaRenderBuffer>;
using namespace std::literals::chrono_literals;

struct TextureData {
    TextureData(const Texture& tex, const Texture& depth_tex, const RenderBuffer& buf, int width, int heigth)
    : surface_texture(tex), depth_texture(depth_tex), render_buffer(buf), width(width), height(height) {
    }
    ~TextureData(){
        surface_texture.reset();
        depth_texture.reset();
        render_buffer.reset();
    };
    Texture surface_texture;
    Texture depth_texture;
    RenderBuffer render_buffer;
    int width;
    int height;
};
static std::shared_ptr<ngp::Testbed> testbed = nullptr;
static std::unordered_map<GLuint, std::shared_ptr<TextureData>> textures;

// flags
bool graphics_initialized = false;
bool use_dlss = false;
bool use_depth = false;
static int _width;
static int _height;
static int _prev_width;
static int _prev_height;
static GLuint leftHandle;
static GLuint rightHandle;
static GLuint leftHandleDepth  = 0;
static GLuint rightHandleDepth = 0;
GLuint* left_handles;
GLuint* right_handles;
float zero_matrix[12] = {0,0,0,0,0,0,0,0,0,0,0,0};
float* view_matrix_left;
float* view_matrix_right;

extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_set_initialize_values(const char* scene, const char* snapshot, bool dlss, bool _use_depth, int width, int height){

    use_dlss = dlss;
    _width = width;
    _height = height;
    _prev_width = width;
    _prev_height = height;
    use_depth = _use_depth;
    view_matrix_left = zero_matrix;
    view_matrix_right = zero_matrix;

    testbed = std::make_shared<ngp::Testbed>(
        ngp::ETestbedMode::Nerf,
		scene
    );

    if (snapshot) {
        testbed->load_snapshot(
			snapshot
        );
    }
    
    // performance setting recommended by the original instant-ngp
    testbed->m_background_color = {0.0f, 0.0f, 0.0f, 0.0f};
    // performance setting recommended by the original instant-ngp
    testbed->m_nerf.render_min_transmittance = 0.2f;

    tlog::info() << "instant ngp testbed created" ;

};

extern "C" bool UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_get_graphics_init_state(){
    return graphics_initialized;
}
// this needs to happen in the render thread
extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_initialize_graphics() { 
    if(graphics_initialized){
        tlog::info() << "graphics already initialized" ;
        return;
    }

	if (!glfwInit()) {
		std::cout << "Could not initialize glfw" << std::endl;
	}
    if (!gl3wInit()) {
        std::cout << "Could not initialize gl3w" << std::endl;
	}

#ifdef NGP_VULKAN
    if (use_dlss) { 
        try {
            ngp::vulkan_and_ngx_init();
        }
        catch (std::runtime_error exception) {
            use_dlss= false;
            std::cout << "Could not initialize vulkan" << std::endl;
        }
    }
#else
    use_dlss = false;
#endif
    
    graphics_initialized = true;
    tlog::info() << "graphics initialized" ;

}

extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_deinit_ngx_vulkan(){
#ifdef NGP_VULKAN    
    if (use_dlss) { 
        ngp::vulkan_and_ngx_destroy();
        use_dlss = false;
    }
#endif
}

extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_deinitialize() {
    textures.clear();
    
// #ifdef NGP_VULKAN    
//     if (use_dlss) { 
//         ngp::vulkan_and_ngx_destroy();
//         use_dlss = false;
//     }
// #endif
    glBindTexture(GL_TEXTURE_2D, 0);
    glDeleteTextures(1, &leftHandleDepth);
    glDeleteTextures(1, &rightHandleDepth);

    testbed.reset();
    glfwTerminate();
    leftHandle = 0;
    rightHandle = 0;
    leftHandleDepth  =  0;
    rightHandleDepth =  0;
    view_matrix_left = NULL;
    view_matrix_right = NULL;
    graphics_initialized = false;
    use_dlss = false;
    tlog::info() << "instant ngp testbed deinitialized" ;

}

static GLuint* UNITY_INTERFACE_API unity_nerf_create_texture(int width, int height) {
    
    if (!testbed){
        tlog::info() << "testbed not found!!" ;
        return 0;
    }
    GLuint* textureHandles = new GLuint[2];
    auto texture   = std::make_shared<ngp::GLTexture>();
    auto depth_tex = std::make_shared<ngp::GLTexture>();
    auto buffer    = std::make_shared<ngp::CudaRenderBuffer>(texture, depth_tex);
    Eigen::Vector2i render_res { width, height }; 

#if defined(NGP_VULKAN)
    if (use_dlss) {
        buffer->enable_dlss({ width, height });
        // buffer->resize({ width, height });

        Eigen::Vector2i texture_res { width, height };
        render_res = buffer->in_resolution();
        if (render_res.isZero()) {
            render_res = texture_res / 16;
        } else {
            render_res = render_res.cwiseMin(texture_res);
        }

        if (buffer->dlss()) {
            render_res = buffer->dlss()->clamp_resolution(render_res);
        }

        //buffer->set_dlss_sharpening(1.0);
    }
    else{ buffer->disable_dlss();}
#endif

    buffer->resize(render_res);

    GLuint handle = texture->texture();
    GLuint handle_depth = depth_tex->texture();

    textures[handle] = std::make_shared<TextureData>(
        texture,
        depth_tex,
        buffer,
        width,
        height
    );

    textureHandles[0] = handle;
    textureHandles[1] = handle_depth;
   // tlog::info() << "GLTexture handle" << handle ;
    return textureHandles;
}


void UNITY_INTERFACE_API unity_nerf_update_texture() {


    if (!testbed){
        tlog::error() << "testbed not found" ;
        return;
    }

    auto left  = textures.find(leftHandle);
    auto right = textures.find(rightHandle);

    if (left == std::end(textures)) {
        tlog::error() << "left texture handle not found" ;
        return;
    }
    if (right == std::end(textures)) {
        tlog::error() << "right texture handle not found" ;
        return;
    }

    Eigen::Matrix<float, 3, 4> camera_left {view_matrix_left};
    Eigen::Matrix<float, 3, 4> camera_right {view_matrix_right};

    RenderBuffer render_buffer_left  = left->second->render_buffer;
    RenderBuffer render_buffer_right = right->second->render_buffer;
    Eigen::Vector2i texture_res { _width, _height };
    Eigen::Vector2i dlss_res { _width, _height };

    {
        // update render latency
        {
        // frame time => we don't need this in instant ngp, since this needs to be calculated in Unity
            auto now = std::chrono::steady_clock::now();
            auto elapsed = now - testbed->m_last_frame_time_point;
            testbed->m_last_frame_time_point = now;
            testbed->m_render_ms.update(std::chrono::duration<float, std::milli>(elapsed).count());
        // auto start = std::chrono::steady_clock::now();
        // tcnn::ScopeGuard timing_guard{[&]() {
        //     testbed->m_render_ms.update(std::chrono::duration<float, std::milli>(std::chrono::steady_clock::now()-start).count());
        // }};
        }
        // implement dynamic resolution here
        if(_prev_width != _width || _prev_height!= _height){
            
            dlss_res = render_buffer_left->in_resolution();
            if (dlss_res.isZero()) {
                dlss_res = texture_res / 16;
            } else {
                dlss_res = dlss_res.cwiseMin(texture_res);
            }
            Eigen::Vector2i dlss_res2 {dlss_res.x(), dlss_res.y() };

            if (render_buffer_left->dlss()) {

			    dlss_res = render_buffer_left ->dlss()->clamp_resolution(dlss_res );
                dlss_res = render_buffer_right->dlss()->clamp_resolution(dlss_res2);
		    }
            render_buffer_left ->resize(dlss_res);
            render_buffer_right->resize(dlss_res);
        }


        testbed->render_frame(camera_left,//testbed->m_camera,
                            camera_left,//testbed->m_camera,
                            Eigen::Vector4f::Zero(),
                            *render_buffer_left.get(),
                            true);

        testbed->render_frame(camera_right,//testbed->m_camera,
                            camera_right,//testbed->m_camera,
                            Eigen::Vector4f::Zero(),
                            *render_buffer_right.get(),
                            true);

        if(use_depth){

            render_buffer_left -> render_depth(ngp::EColorSpace::SRGB, testbed->m_stream.get());
            render_buffer_right-> render_depth(ngp::EColorSpace::SRGB, testbed->m_stream.get());
        }

        render_buffer_left ->reset_accumulation();
        render_buffer_right->reset_accumulation();
    
    }
    _prev_width  = _width;
    _prev_height = _height;

    left->second->width  = _width;
    left->second->height = _height;
    right->second->width = _width;
    right->second->height= _height;

    testbed->reset_accumulation(true);

}
extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_save_snapshot(const char* filename){
    if (!testbed){
        tlog::error() << "testbed not found" ;
        return;
    }
    tlog::info() << "testbed not found" ;

    testbed->save_snapshot(filename, false);
};


extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_update_resolution(int width, int height){
    
    // this function must be called in Unity before initiate the "unity_nerf_update_texture" event.
    
    if (!testbed){
        tlog::error() << "testbed not found" ;
        return;
    }

    _width = width;
    _height = height;

}

extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_destroy_texture(GLuint handle) {

    if (!testbed)
        return;
    auto found = textures.find(handle); 
    if (found == std::end(textures)) {
        return;
    } 

    found->second->render_buffer->reset_accumulation();
    found->second->render_buffer.reset();
    found->second->surface_texture.reset();
    found->second->depth_texture.reset();
    found->second.reset();
    tlog::info() << "GLTexture and render buffer destroyed" ;

}

// remove certain render volume
extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_set_all_density_grid_empty(){
    if (!testbed)
    return;
    testbed->mark_all_density_grid_empty(testbed->m_stream.get());
}

// remove certain render volume
extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_reveal_all_masked_density(){
    if (!testbed)
    return;
    testbed->reveal_all_masked_density(testbed->m_stream.get());
}

// remove certain render volume
extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_mark_density_grid_empty(float* pos, float scale){
    if (!testbed)
    return;
    Eigen::Vector3<float> remove_pos {pos};
    testbed->mark_density_grid_in_sphere_empty(remove_pos, scale*0.05f, testbed->m_stream.get());
}
// erase density grid defined by a box
extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_empty_density_grid_in_box(float* pos, float box_width, float box_height, float box_length, float* R){
    if (!testbed)
    return;
    Eigen::Vector3<float> remove_pos {pos};
    Eigen::Matrix<float, 3, 3> _R {R};
    testbed->erase_volume_density_in_box(remove_pos, box_width, box_height, box_length, _R, testbed->m_stream.get());
}
// reveal certain render area
extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_reveal_density_grid_area(float* pos, float scale){
    if (!testbed)
    return;
    Eigen::Vector3<float> remove_pos {pos};
    testbed->reveal_volume_density_in_sphere(remove_pos, scale*0.05f, testbed->m_stream.get());
}
// reveal density grid defined by a box
extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_reveal_density_grid_in_box(float* pos, float box_width, float box_height, float box_length, float* R){
    if (!testbed)
    return;
    Eigen::Vector3<float> remove_pos {pos};
    Eigen::Matrix<float, 3, 3> _R {R};
    testbed->reveal_volume_density_in_box(remove_pos, box_width, box_height, box_length, _R, testbed->m_stream.get());
}
// utility functions

extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_reset_camera(){
    if (!testbed)
    return;
    testbed->reset_camera();
}

extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_set_render_fov(float val){
    if (!testbed)
        return;
    testbed->set_fov(val);

};

extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_set_scale(float val){
    if (!testbed)
        return;

    testbed->set_scale(val);
};

extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_set_look_at(float val[]){
    if (!testbed)
        return;
    testbed->set_look_at({val[0],val[1],val[2]});
};

extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_get_look_at(float val[]){
    if (!testbed)
        return;
    float* v = testbed->look_at().data();
    for(int i=0; i<3; i++){
        val[i] = v[i];
    }
}

extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_get_view_dir(float val[]){
    if (!testbed)
        return;
    float* v = testbed->view_dir().data();
    for(int i=0; i<3; i++){
        val[i] = v[i];
    }
};

extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_set_view_dir(float val[]){
    if (!testbed)
        return;
    testbed->set_view_dir({val[0], val[1], val[2]});
};


extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_update_aabb_crop(float* min_vec, float* max_vec){

    if (!testbed)
        return;

    Eigen::Vector3f min_aabb {min_vec};
    Eigen::Vector3f max_aabb {max_vec};
    
    testbed->m_render_aabb = ngp::BoundingBox(min_aabb, max_aabb);

}

extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_update_aabb(float* min_vec, float* max_vec){
    // this update the generate scene where the user can see.
    if (!testbed)
        return;

    Eigen::Vector3f min_aabb {min_vec};
    Eigen::Vector3f max_aabb {max_vec};
    
    testbed->m_aabb = ngp::BoundingBox(min_aabb, max_aabb);
}

// how to pass array ptr to unity:  https://bravenewmethod.com/2017/10/30/unity-c-native-plugin-examples/
// how to visualize the unit cube in unity and manipulate it: 
// look at line 1239-1264 in testbed.cu for the instant-ngp implementation
extern "C" UNITY_INTERFACE_EXPORT float* UNITY_INTERFACE_API unity_nerf_get_render_aabb_min(){

    if (!testbed)
        return NULL;
    float* min = testbed->m_render_aabb.min.data();
    return min;

} 

extern "C" UNITY_INTERFACE_EXPORT float* UNITY_INTERFACE_API unity_nerf_get_render_aabb_max(){
    
    if (!testbed)
        return NULL;
    float* max = testbed->m_render_aabb.max.data();
    return max;

};

extern "C" UNITY_INTERFACE_EXPORT float* UNITY_INTERFACE_API unity_nerf_get_aabb_min(){
    if(!testbed)
        return NULL;
    float* min = testbed->m_aabb.min.data();
    return min;
};

extern "C" UNITY_INTERFACE_EXPORT float* UNITY_INTERFACE_API unity_nerf_get_aabb_max(){
    if(!testbed)
        return NULL;

    float* max = testbed->m_aabb.max.data();
    return max;
};


extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_get_render_aabb_to_local(float arr[]){

    if (!testbed)
        return;
    float* aabb_to_local = testbed->m_render_aabb_to_local.data();
    for(int i =0; i< 9; i++){
        arr[i] = aabb_to_local[i];
    }
} 

extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_get_crop_box_transform(float arr[]){
    if (!testbed)
        return;
    // eigen is column major 
    float* ptr2 = testbed ->crop_box(false).data();
    for (int i = 0; i < testbed ->crop_box(false).size(); i++) {
        arr[i] = ptr2[i];
    }
}

extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_set_crop_box_transform(float* cropbox){
    if (!testbed)
        return;
    Eigen::Matrix<float, 3, 4> m { cropbox };
    testbed->set_crop_box(m, false);
};

const int INIT_EVENT = 0x0001;
const int DRAW_EVENT = 0x0002;
const int DEINIT_EVENT = 0x0003;
const int CREATE_TEX = 0x0004;
const int DEINIT_VULKAN = 0x0005;

static void UNITY_INTERFACE_API unity_nerf_run_on_render_thread(int eventID)
{

    switch (eventID)
    {

        case INIT_EVENT:
            
            unity_nerf_initialize_graphics();

            break;

        case CREATE_TEX:

            left_handles  = unity_nerf_create_texture(_width, _height);
            right_handles = unity_nerf_create_texture(_width, _height);
            
            leftHandle  = left_handles[0];
            rightHandle = right_handles[0];
            leftHandleDepth  = left_handles[1];
            rightHandleDepth = right_handles[1];

            delete[] left_handles;
            delete[] right_handles;

            break;

        case DRAW_EVENT:
            
            unity_nerf_update_texture();

            break;


        case DEINIT_EVENT:

            unity_nerf_destroy_texture(leftHandle);
            unity_nerf_destroy_texture(rightHandle);
            unity_nerf_deinitialize();
            break;

        case DEINIT_VULKAN:

            unity_nerf_deinit_ngx_vulkan();
            break;

    }
}


// --------------------------------------------------------------------------
// GetRenderEventFunc, an example function we export which is used to get a rendering event callback function.

extern "C" UnityRenderingEvent UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API GetRenderEventFunc(){

	return unity_nerf_run_on_render_thread;
}

extern "C" GLuint UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_get_left_handle(){
    
    return leftHandle;
}

extern "C" GLuint UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_get_right_handle(){

    return rightHandle;
}

extern "C" GLuint UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_get_left_depth_handle(){
    
    return leftHandleDepth;
}

extern "C" GLuint UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_get_right_depth_handle(){

    return rightHandleDepth;
}

extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_update_stereo_view_matrix(float* left, float* right){
    view_matrix_left = left;
    view_matrix_right = right;
}

extern "C" float UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unity_nerf_get_render_ms(){
    if (!testbed){
        tlog::error() << "testbed not found" ;
        return 0;
    }
    return testbed->m_render_ms.ema_val();
}